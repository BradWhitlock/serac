#include "hip/hip_runtime.h"
#include "mfem.hpp"

#include "mfem_PA_kernels_h1.hpp"
#include "mfem_PA_kernels_hcurl.hpp"

#include "axom/core/utilities/Timer.hpp"

#include "serac/infrastructure/accelerator.hpp"
#include "serac/numerics/functional/tensor.hpp"
#include "serac/numerics/functional/quadrature.hpp"
#include "serac/numerics/functional/finite_element.hpp"
#include "serac/numerics/functional/tuple_arithmetic.hpp"
#include "serac/numerics/functional/integral_utilities.hpp"

#include <type_traits>

namespace serac {

template <Geometry g, int Q>
struct GaussLegendreRule;

template <int Q>
struct GaussLegendreRule<Geometry::Quadrilateral, Q> {
  static constexpr auto points_1D  = GaussLegendreNodes<Q>();
  static constexpr auto weights_1D = GaussLegendreWeights<Q>();

  static constexpr double weight(int qx, int qy) { return weights_1D[qx] * weights_1D[qy]; }

  static constexpr int size() { return Q * Q; }
};

template <int Q>
struct GaussLegendreRule<Geometry::Hexahedron, Q> {
  static constexpr auto points_1D  = GaussLegendreNodes<Q>();
  static constexpr auto weights_1D = GaussLegendreWeights<Q>();

  static constexpr double weight(int qx, int qy, int qz) { return weights_1D[qx] * weights_1D[qy] * weights_1D[qz]; }

  static constexpr int size() { return Q * Q * Q; }
};

template <Geometry g, typename test, typename trial, int Q, typename lambda>
__global__ void reference_cuda_kernel(mfem::DeviceTensor<2, const double> u, mfem::DeviceTensor<2, double> r,
                                      mfem::DeviceTensor<4, const double> J, size_t num_elements, lambda qf)
{
  using test_element          = finite_element<g, test>;
  using trial_element         = finite_element<g, trial>;
  using element_residual_type = typename test_element::residual_type;
  static constexpr auto rule  = GaussQuadratureRule<g, Q>();
  static constexpr int  dim   = dimension_of(g);

  const int grid_stride = blockDim.x * gridDim.x;

  for (int qe = blockIdx.x * blockDim.x + threadIdx.x; qe < num_elements * rule.size(); qe += grid_stride) {
    int e = qe / rule.size();
    int q = qe % rule.size();

    auto u_elem = detail::Load<trial_element>(u, e);

    element_residual_type r_elem{};

    auto   xi  = rule.points[q];
    auto   dxi = rule.weights[q];
    auto   J_q = make_tensor<dim, dim>([&](int i, int j) { return J(q, i, j, e); });
    double dx  = det(J_q) * dxi;

    auto arg = domain_integral::Preprocess<trial_element>(u_elem, xi, J_q);

    auto qf_output = qf(arg);

    r_elem += domain_integral::Postprocess<test_element>(qf_output, xi, J_q) * dx;

    detail::Add(r, r_elem, e);
  }
}

template <int dim, int q>
__device__ auto load_jacobian(const tensor<double, dim, dim, q, q>& J)
{
  int tidx = threadIdx.x % q;
  int tidy = threadIdx.x / q;

  tensor<double, dim, dim> J_q;
  for (int i = 0; i < dim; i++) {
    for (int j = 0; j < dim; j++) {
      J_q[i][j] = J(j, i, tidy, tidx);
    }
  }
  return J_q;
}

template <int dim, int q>
__device__ auto load_jacobian(const tensor<double, dim, dim, q, q, q>& J)
{
  int tidx = threadIdx.x % q;
  int tidy = (threadIdx.x % (q * q)) / q;
  int tidz = threadIdx.x / (q * q);

  tensor<double, dim, dim> J_q;
  for (int i = 0; i < dim; i++) {
    for (int j = 0; j < dim; j++) {
      for (int k = 0; k < dim; k++) {
        J_q[i][j] = J(j, i, tidz, tidy, tidx);
      }
    }
  }
  return J_q;
}

template <typename dof_type>
__device__ void load(const dof_type& source, dof_type& destination)
{
  constexpr int ndof    = sizeof(dof_type) / sizeof(double);
  const double* src_ptr = reinterpret_cast<const double*>(&source);
  double*       dst_ptr = reinterpret_cast<double*>(&destination);
  for (int i = threadIdx.x; i < ndof; i += blockDim.x) {
    dst_ptr[i] = src_ptr[i];
  }
}

template <Geometry g, typename test, typename trial, int q, typename lambda>
__global__ void batched_cuda_kernel(const double* inputs, double* outputs, const double* jacobians,
                                    TensorProductQuadratureRule<q> rule, size_t num_elements, lambda material)
{
  using test_element  = finite_element<g, test>;
  using trial_element = finite_element<g, trial>;

  auto u = reinterpret_cast<const typename trial_element::dof_type*>(inputs);
  auto r = reinterpret_cast<typename test_element::dof_type*>(outputs);
  auto J = reinterpret_cast<const typename batched_jacobian<g, q>::type*>(jacobians);

  int e = threadIdx.y + elements_per_block<g>(q) * blockIdx.x;

  if (e < num_elements) {
    __shared__ union {
      typename trial_element::cache_type<q> trial_cache;
      typename test_element::cache_type<q>  test_cache;
    } shared[elements_per_block<g>(q)];

    // load the element values for this element
    __shared__ typename trial_element::dof_type u_elem[elements_per_block<g>(q)];
    load(u[e], u_elem[threadIdx.y]);

    // and load the jacobian for this thread's quadrature point
    auto J_q = load_jacobian(J[e]);

    // interpolate each quadrature point's value
    auto stimulus = trial_element::interpolate(u_elem[threadIdx.y], J_q, rule, shared[threadIdx.y].trial_cache);

    // evaluate the material response at each quadrature point
    auto response = material(stimulus);

    // integrate the material response against the test-space basis functions
    test_element::integrate(response, J_q, rule, shared[threadIdx.y].test_cache, r[e]);
  }
}

}  // namespace serac

namespace compiler {
static void please_do_not_optimize_away([[maybe_unused]] void* p) { asm volatile("" : : "g"(p) : "memory"); }
}  // namespace compiler

template <typename lambda>
auto time(lambda&& f)
{
  axom::utilities::Timer stopwatch;
  stopwatch.start();
  f();
  stopwatch.stop();
  return stopwatch.elapsed();
}

struct MassAndDiffusionQFunction {
  template <typename T>
  SERAC_HOST_DEVICE auto operator()(T input)
  {
    auto [u, du_dx] = input;
    auto source     = rho * u;
    auto flux       = k * du_dx;
    return serac::tuple{source, flux};
  }

  double rho;
  double k;
};

constexpr double k   = 2.0;
constexpr double rho = 2.0;

constexpr MassAndDiffusionQFunction qfunc{rho, k};

template <int p, int q>
void h1_h1_test_2D(int num_elements, int num_runs)
{
  using serac::Geometry;
  using serac::H1;

  constexpr int n   = p + 1;
  constexpr int dim = 2;

  using test  = H1<p>;
  using trial = H1<p>;

  std::default_random_engine             generator;
  std::uniform_real_distribution<double> distribution(-1.0, 1.0);

  mfem::Vector U1D(num_elements * n * n);
  mfem::Vector R1D(num_elements * n * n);
  mfem::Vector J1D(num_elements * dim * dim * q * q);
  mfem::Vector rho_dv_1D(num_elements * q * q);
  mfem::Vector k_invJ_invJT_dv_1D(num_elements * dim * dim * q * q);

  auto U               = mfem::Reshape(U1D.HostReadWrite(), n, n, num_elements);
  auto J               = mfem::Reshape(J1D.HostReadWrite(), q * q, dim, dim, num_elements);
  auto rho_dv          = mfem::Reshape(rho_dv_1D.HostReadWrite(), q * q, num_elements);
  auto k_invJ_invJT_dv = mfem::Reshape(k_invJ_invJT_dv_1D.HostReadWrite(), q * q, dim, dim, num_elements);

  serac::GaussLegendreRule<Geometry::Quadrilateral, q> rule;

  for (int e = 0; e < num_elements; e++) {
    for (int ix = 0; ix < n; ix++) {
      for (int iy = 0; iy < n; iy++) {
        U(iy, ix, e) = 0.1 * distribution(generator);
      }
    }

    for (int i = 0; i < q * q; i++) {
      serac::tensor<double, dim, dim> J_q{};

      for (int r = 0; r < dim; r++) {
        for (int c = 0; c < dim; c++) {
          J(i, r, c, e) = J_q[r][c] = (r == c) + 0.1 * distribution(generator);
        }
      }

      int qx = i % q;
      int qy = i / q;

      double qweight    = rule.weight(qx, qy);
      auto   invJ_invJT = dot(inv(J_q), transpose(inv(J_q)));
      double dv         = det(J_q) * qweight;

      rho_dv(i, e) = rho * dv;
      for (int r = 0; r < dim; r++) {
        for (int c = 0; c < dim; c++) {
          k_invJ_invJT_dv(i, r, c, e) = k * invJ_invJT[r][c] * dv;
        }
      }
    }
  }

  {
    R1D = 0.0;

    mfem::DeviceTensor<2, const double> u_d       = mfem::Reshape(U1D.Read(), n * n, num_elements);
    mfem::DeviceTensor<2, double>       r_d       = mfem::Reshape(R1D.ReadWrite(), n * n, num_elements);
    mfem::DeviceTensor<4, const double> J_d       = mfem::Reshape(J1D.Read(), q * q, dim, dim, num_elements);
    int                                 blocksize = 128;
    int                                 gridsize  = (num_elements * q * q + blocksize - 1) / blocksize;
    double                              runtime   = time([&]() {
      for (int i = 0; i < num_runs; i++) {
        serac::reference_cuda_kernel<Geometry::Quadrilateral, test, trial, q>
            <<<gridsize, blocksize>>>(u_d, r_d, J_d, num_elements, qfunc);
        compiler::please_do_not_optimize_away(&R1D);
      }
      hipDeviceSynchronize();
    });
    std::cout << "average reference kernel time: " << runtime / num_runs << std::endl;
  }
  auto answer_reference = R1D;

  {
    R1D                = 0.0;
    auto          rule = serac::MakeGaussLegendreRule<Geometry::Quadrilateral, q>();
    constexpr int epb  = serac::elements_per_block<Geometry::Quadrilateral>(q);
    dim3          blocksize{q * q, epb, 1};
    int           gridsize = (num_elements + epb - 1) / epb;
    double        runtime  = time([&]() {
      for (int i = 0; i < num_runs; i++) {
        serac::batched_cuda_kernel<Geometry::Quadrilateral, test, trial, q>
            <<<gridsize, blocksize>>>(U1D.Read(), R1D.ReadWrite(), J1D.Read(), rule, num_elements, qfunc);
        compiler::please_do_not_optimize_away(&R1D);
      }
      hipDeviceSynchronize();
    });
    std::cout << "average cpu batched kernel time: " << runtime / num_runs << std::endl;
  }
  auto answer_cpu_batched_kernel = R1D;
  auto error                     = answer_reference;
  error -= answer_cpu_batched_kernel;
  double relative_error = error.Norml2() / answer_reference.Norml2();
  std::cout << "error: " << relative_error << std::endl;

#if 0
  {
    R1D                           = 0.0;
    bool                symmetric = false;
    mfem::Array<double> b_(n * q);
    mfem::Array<double> bt_(n * q);
    mfem::Array<double> g_(n * q);
    mfem::Array<double> gt_(n * q);
    auto                B  = mfem::Reshape(b_.ReadWrite(), q, n);
    auto                Bt = mfem::Reshape(bt_.ReadWrite(), n, q);

    auto G  = mfem::Reshape(g_.ReadWrite(), q, n);
    auto Gt = mfem::Reshape(gt_.ReadWrite(), n, q);

    for (int i = 0; i < q; i++) {
      auto value      = serac::GaussLobattoInterpolation<n>(rule.points_1D[i]);
      auto derivative = serac::GaussLobattoInterpolationDerivative<n>(rule.points_1D[i]);

      for (int j = 0; j < n; j++) {
        Bt(j, i) = B(i, j) = value[j];
        Gt(j, i) = G(i, j) = derivative[j];
      }
    }

    double mass_runtime = time([&]() {
                            for (int i = 0; i < num_runs; i++) {
                              mfem::SmemPAMassApply3D<n, q>(num_elements, b_, bt_, rho_dv_1D, U1D, R1D);
                              compiler::please_do_not_optimize_away(&R1D);
                            }
                          }) /
                          n;
    std::cout << "average mfem mass kernel time: " << mass_runtime / num_runs << std::endl;

    double diffusion_runtime =
        time([&]() {
          for (int i = 0; i < num_runs; i++) {
            mfem::SmemPADiffusionApply3D<n, q>(num_elements, symmetric = false, b_, g_, k_invJ_invJT_dv_1D, U1D, R1D);
            compiler::please_do_not_optimize_away(&R1D);
          }
        }) /
        n;
    std::cout << "average mfem diffusion kernel time: " << diffusion_runtime / num_runs << std::endl;

    std::cout << "average mfem combined kernel time: " << (mass_runtime + diffusion_runtime) / num_runs << std::endl;
  }
  auto answer_mfem = R1D;
  error            = answer_reference;
  error -= answer_mfem;
  relative_error = error.Norml2() / answer_reference.Norml2();
  std::cout << "error: " << relative_error << std::endl;
#endif
}

template <int p, int q>
void h1_h1_test_3D(int num_elements, int num_runs)
{
  using serac::Geometry;
  using serac::H1;

  constexpr int n   = p + 1;
  constexpr int dim = 3;

  using test  = H1<p>;
  using trial = H1<p>;

  std::default_random_engine             generator;
  std::uniform_real_distribution<double> distribution(-1.0, 1.0);

  mfem::Vector U1D(num_elements * n * n * n);
  mfem::Vector R1D(num_elements * n * n * n);
  mfem::Vector J1D(num_elements * dim * dim * q * q * q);
  mfem::Vector rho_dv_1D(num_elements * q * q * q);
  mfem::Vector k_invJ_invJT_dv_1D(num_elements * dim * dim * q * q * q);

  auto U               = mfem::Reshape(U1D.HostReadWrite(), n, n, n, num_elements);
  auto J               = mfem::Reshape(J1D.HostReadWrite(), q * q * q, dim, dim, num_elements);
  auto rho_dv          = mfem::Reshape(rho_dv_1D.HostReadWrite(), q * q * q, num_elements);
  auto k_invJ_invJT_dv = mfem::Reshape(k_invJ_invJT_dv_1D.HostReadWrite(), q * q * q, dim, dim, num_elements);

  serac::GaussLegendreRule<Geometry::Hexahedron, q> rule;

  for (int e = 0; e < num_elements; e++) {
    for (int ix = 0; ix < n; ix++) {
      for (int iy = 0; iy < n; iy++) {
        for (int iz = 0; iz < n; iz++) {
          U(iz, iy, ix, e) = 0.1 * distribution(generator);
        }
      }
    }

    for (int i = 0; i < q * q * q; i++) {
      serac::tensor<double, dim, dim> J_q{};

      for (int r = 0; r < dim; r++) {
        for (int c = 0; c < dim; c++) {
          J(i, r, c, e) = J_q[r][c] = (r == c) + 0.1 * distribution(generator);
        }
      }

      int qx = i % q;
      int qy = (i % (q * q)) / q;
      int qz = i / (q * q);

      double qweight    = rule.weight(qx, qy, qz);
      auto   invJ_invJT = dot(inv(J_q), transpose(inv(J_q)));
      double dv         = det(J_q) * qweight;

      rho_dv(i, e) = rho * dv;
      for (int r = 0; r < dim; r++) {
        for (int c = 0; c < dim; c++) {
          k_invJ_invJT_dv(i, r, c, e) = k * invJ_invJT[r][c] * dv;
        }
      }
    }
  }

  {
    R1D = 0.0;

    mfem::DeviceTensor<2, const double> u_d       = mfem::Reshape(U1D.Read(), n * n * n, num_elements);
    mfem::DeviceTensor<2, double>       r_d       = mfem::Reshape(R1D.ReadWrite(), n * n * n, num_elements);
    mfem::DeviceTensor<4, const double> J_d       = mfem::Reshape(J1D.Read(), q * q * q, dim, dim, num_elements);
    int                                 blocksize = 128;
    int                                 gridsize  = (num_elements * q * q * q + blocksize - 1) / blocksize;
    double                              runtime   = time([&]() {
      for (int i = 0; i < num_runs; i++) {
        serac::reference_cuda_kernel<Geometry::Hexahedron, test, trial, q>
            <<<gridsize, blocksize>>>(u_d, r_d, J_d, num_elements, qfunc);
        compiler::please_do_not_optimize_away(&R1D);
      }
      hipDeviceSynchronize();
    });
    std::cout << "average refernce kernel time: " << runtime / num_runs << std::endl;
  }
  auto answer_reference = R1D;

  {
    R1D                = 0.0;
    auto          rule = serac::MakeGaussLegendreRule<Geometry::Hexahedron, q>();
    constexpr int epb  = serac::elements_per_block<Geometry::Hexahedron>(q);
    dim3          blocksize{q * q * q, epb, 1};
    int           gridsize = (num_elements + epb - 1) / epb;
    double        runtime  = time([&]() {
      for (int i = 0; i < num_runs; i++) {
        serac::batched_cuda_kernel<Geometry::Hexahedron, test, trial, q>
            <<<gridsize, blocksize>>>(U1D.Read(), R1D.ReadWrite(), J1D.Read(), rule, num_elements, qfunc);
        compiler::please_do_not_optimize_away(&R1D);
      }
      hipDeviceSynchronize();
    });
    std::cout << "average batched kernel time: " << runtime / num_runs << std::endl;
  }
  auto error = answer_reference;
  error -= R1D;
  double relative_error = error.Norml2() / answer_reference.Norml2();
  std::cout << "error: " << relative_error << std::endl;

  {
    R1D                           = 0.0;
    bool                symmetric = false;
    mfem::Array<double> b_(n * q);
    mfem::Array<double> bt_(n * q);
    mfem::Array<double> g_(n * q);
    mfem::Array<double> gt_(n * q);
    auto                B  = mfem::Reshape(b_.HostReadWrite(), q, n);
    auto                Bt = mfem::Reshape(bt_.HostReadWrite(), n, q);

    auto G  = mfem::Reshape(g_.HostReadWrite(), q, n);
    auto Gt = mfem::Reshape(gt_.HostReadWrite(), n, q);

    for (int i = 0; i < q; i++) {
      auto value      = serac::GaussLobattoInterpolation<n>(rule.points_1D[i]);
      auto derivative = serac::GaussLobattoInterpolationDerivative<n>(rule.points_1D[i]);

      for (int j = 0; j < n; j++) {
        Bt(j, i) = B(i, j) = value[j];
        Gt(j, i) = G(i, j) = derivative[j];
      }
    }

    double mass_runtime = time([&]() {
      for (int i = 0; i < num_runs; i++) {
        mfem::SmemPAMassApply3D<n, q>(num_elements, b_, bt_, rho_dv_1D, U1D, R1D);
        compiler::please_do_not_optimize_away(&R1D);
      }
    });
    std::cout << "average mfem mass kernel time: " << mass_runtime / num_runs << std::endl;

    double diffusion_runtime = time([&]() {
      for (int i = 0; i < num_runs; i++) {
        mfem::SmemPADiffusionApply3D<n, q>(num_elements, symmetric = false, b_, g_, k_invJ_invJT_dv_1D, U1D, R1D);
        compiler::please_do_not_optimize_away(&R1D);
      }
    });
    std::cout << "average mfem diffusion kernel time: " << diffusion_runtime / num_runs << std::endl;

    std::cout << "average mfem combined kernel time: " << (mass_runtime + diffusion_runtime) / num_runs << std::endl;
  }
  error = answer_reference;
  error -= R1D;
  relative_error = error.Norml2() / answer_reference.Norml2();
  std::cout << "error: " << relative_error << std::endl;
}

template <int p, int q>
void hcurl_hcurl_test_2D(int num_elements, int num_runs)
{
  using serac::Geometry;
  using serac::Hcurl;

  constexpr int n   = p + 1;
  constexpr int dim = 2;

  using test  = Hcurl<p>;
  using trial = Hcurl<p>;

  using trial_element = serac::finite_element<Geometry::Quadrilateral, trial>;
  using test_element  = serac::finite_element<Geometry::Quadrilateral, test>;

  std::default_random_engine             generator;
  std::uniform_real_distribution<double> distribution(-1.0, 1.0);

  mfem::Vector U1D(num_elements * trial_element::ndof);
  mfem::Vector R1D(num_elements * test_element::ndof);
  mfem::Vector J1D(num_elements * dim * dim * q * q);

  auto U = mfem::Reshape(U1D.HostReadWrite(), trial_element::ndof, num_elements);
  auto J = mfem::Reshape(J1D.HostReadWrite(), q * q, dim, dim, num_elements);

  for (int e = 0; e < num_elements; e++) {
    for (int i = 0; i < trial_element::ndof; i++) {
      U(i, e) = 0.1 * distribution(generator);
    }

    for (int i = 0; i < q * q; i++) {
      serac::tensor<double, dim, dim> J_q{};

      for (int r = 0; r < dim; r++) {
        for (int c = 0; c < dim; c++) {
          J(i, r, c, e) = J_q[r][c] = (r == c) + 0.1 * distribution(generator);
        }
      }
    }
  }

  {
    R1D = 0.0;

    mfem::DeviceTensor<2, const double> u_d       = mfem::Reshape(U1D.Read(), trial_element::ndof, num_elements);
    mfem::DeviceTensor<2, double>       r_d       = mfem::Reshape(R1D.ReadWrite(), test_element::ndof, num_elements);
    mfem::DeviceTensor<4, const double> J_d       = mfem::Reshape(J1D.Read(), q * q, dim, dim, num_elements);
    int                                 blocksize = 128;
    int                                 gridsize  = (num_elements * q * q + blocksize - 1) / blocksize;
    double                              runtime   = time([&]() {
      for (int i = 0; i < num_runs; i++) {
        serac::reference_cuda_kernel<Geometry::Quadrilateral, test, trial, q>
            <<<gridsize, blocksize>>>(u_d, r_d, J_d, num_elements, qfunc);
        compiler::please_do_not_optimize_away(&R1D);
      }
      hipDeviceSynchronize();
    });
    std::cout << "average refernce kernel time: " << runtime / num_runs << std::endl;
  }
  auto answer_reference = R1D;

  {
    R1D = 0.0;

    auto          rule = serac::MakeGaussLegendreRule<Geometry::Quadrilateral, q>();
    constexpr int epb  = serac::elements_per_block<Geometry::Quadrilateral>(q);
    dim3          blocksize{q * q, epb, 1};
    int           gridsize = (num_elements + epb - 1) / epb;
    double        runtime  = time([&]() {
      for (int i = 0; i < num_runs; i++) {
        serac::batched_cuda_kernel<Geometry::Quadrilateral, test, trial, q>
            <<<gridsize, blocksize>>>(U1D.Read(), R1D.ReadWrite(), J1D.Read(), rule, num_elements, qfunc);
        compiler::please_do_not_optimize_away(&R1D);
      }
      hipDeviceSynchronize();
    });
    std::cout << "average batched kernel time: " << runtime / num_runs << std::endl;
  }
  auto error = answer_reference;
  error -= R1D;
  double relative_error = error.Norml2() / answer_reference.Norml2();
  std::cout << "error: " << relative_error << std::endl;

#if 0
  {
    R1D            = 0.0;
    bool symmetric = false;

    // I think the "o" and "c" are supposed to be short for
    // "open" and "closed", referring to placing interpolation
    // nodes at the gauss-legendre, and gauss-lobatto points (respectively)
    mfem::Array<double> bo_((n - 1) * q);
    mfem::Array<double> bc_(n * q);
    mfem::Array<double> bot_((n - 1) * q);
    mfem::Array<double> bct_(n * q);
    mfem::Array<double> gc_(n * q);
    mfem::Array<double> gct_(n * q);

    auto Bo  = mfem::Reshape(bo_.ReadWrite(), q, n - 1);
    auto Bc  = mfem::Reshape(bc_.ReadWrite(), q, n);
    auto Bot = mfem::Reshape(bot_.ReadWrite(), n - 1, q);
    auto Bct = mfem::Reshape(bct_.ReadWrite(), n, q);
    auto Gc  = mfem::Reshape(gc_.ReadWrite(), q, n);
    auto Gct = mfem::Reshape(gct_.ReadWrite(), n, q);

    for (int i = 0; i < q; i++) {
      auto lobatto_value      = serac::GaussLobattoInterpolation<n>(rule.points_1D[i]);
      auto lobatto_derivative = serac::GaussLobattoInterpolationDerivative<n>(rule.points_1D[i]);

      for (int j = 0; j < n; j++) {
        Bct(j, i) = Bc(i, j) = lobatto_value[j];
        Gct(j, i) = Gc(i, j) = lobatto_derivative[j];
      }

      auto legendre_value = serac::GaussLegendreInterpolation<n - 1>(rule.points_1D[i]);
      for (int j = 0; j < n - 1; j++) {
        Bot(j, i) = Bo(i, j) = legendre_value[j];
      }
    }

    double mass_runtime = time([&]() {
                            for (int i = 0; i < num_runs; i++) {
                              mfem::PAHcurlMassApply3D(n, q, num_elements, symmetric = false, bo_, bc_, bot_, bct_,
                                                       rho_invJ_invJT_dv_1D, U1D, R1D);
                              compiler::please_do_not_optimize_away(&R1D);
                            }
                          });
    std::cout << "average mfem mass kernel time: " << mass_runtime / num_runs << std::endl;

    double curlcurl_runtime = time([&]() {
                                 for (int i = 0; i < num_runs; i++) {
                                   mfem::PACurlCurlApply3D<n, q>(n, q, symmetric = false, num_elements, bo_, bc_, bot_,
                                                                 bct_, gc_, gct_, k_JTJ_dv_over_detJsq_1D, U1D, R1D);
                                   compiler::please_do_not_optimize_away(&R1D);
                                 }
                               });
    std::cout << "average mfem curlcurl kernel time: " << curlcurl_runtime / num_runs << std::endl;

    std::cout << "average mfem combined kernel time: " << (mass_runtime + curlcurl_runtime) / num_runs << std::endl;
  }
  auto answer_mfem = R1D;
  error            = answer_reference;
  error -= answer_mfem;
  relative_error = error.Norml2() / answer_reference.Norml2();
  std::cout << "error: " << relative_error << std::endl;
#endif
}

template <int p, int q>
void hcurl_hcurl_test_3D(int num_elements, int num_runs)
{
  using serac::Geometry;
  using serac::Hcurl;

  constexpr int n   = p + 1;
  constexpr int dim = 3;

  using test  = Hcurl<p>;
  using trial = Hcurl<p>;

  using trial_element = serac::finite_element<Geometry::Hexahedron, trial>;
  using test_element  = serac::finite_element<Geometry::Hexahedron, test>;

  std::default_random_engine             generator;
  std::uniform_real_distribution<double> distribution(-1.0, 1.0);

  mfem::Vector U1D(num_elements * trial_element::ndof);
  mfem::Vector R1D(num_elements * test_element::ndof);
  mfem::Vector J1D(num_elements * dim * dim * q * q * q);
  mfem::Vector rho_invJ_invJT_dv_1D(num_elements * dim * dim * q * q * q);
  mfem::Vector k_JTJ_dv_over_detJsq_1D(num_elements * dim * dim * q * q * q);

  auto U                    = mfem::Reshape(U1D.HostReadWrite(), trial_element::ndof, num_elements);
  auto J                    = mfem::Reshape(J1D.HostReadWrite(), q * q * q, dim, dim, num_elements);
  auto rho_invJ_invJT_dv    = mfem::Reshape(rho_invJ_invJT_dv_1D.HostReadWrite(), q * q * q, dim, dim, num_elements);
  auto k_JTJ_dv_over_detJsq = mfem::Reshape(k_JTJ_dv_over_detJsq_1D.HostReadWrite(), q * q * q, dim, dim, num_elements);

  serac::GaussLegendreRule<Geometry::Hexahedron, q> rule;

  for (int e = 0; e < num_elements; e++) {
    for (int i = 0; i < trial_element::ndof; i++) {
      U(i, e) = 0.1 * distribution(generator);
    }

    for (int i = 0; i < q * q * q; i++) {
      serac::tensor<double, dim, dim> J_q{};

      for (int r = 0; r < dim; r++) {
        for (int c = 0; c < dim; c++) {
          J(i, r, c, e) = J_q[r][c] = (r == c) + 0.1 * distribution(generator);
        }
      }

      int qx = i % q;
      int qy = (i % (q * q)) / q;
      int qz = i / (q * q);

      double qweight    = rule.weight(qx, qy, qz);
      auto   JTJ        = dot(transpose(J_q), J_q);
      auto   invJ_invJT = dot(inv(J_q), transpose(inv(J_q)));
      auto   detJ       = det(J_q);
      double dv         = det(J_q) * qweight;

      for (int r = 0; r < dim; r++) {
        for (int c = 0; c < dim; c++) {
          k_JTJ_dv_over_detJsq(i, r, c, e) = k * (JTJ[r][c] / (detJ * detJ)) * dv;
          rho_invJ_invJT_dv(i, r, c, e)    = rho * invJ_invJT[r][c] * dv;
        }
      }
    }
  }

  {
    R1D = 0.0;

    mfem::DeviceTensor<2, const double> u_d       = mfem::Reshape(U1D.Read(), trial_element::ndof, num_elements);
    mfem::DeviceTensor<2, double>       r_d       = mfem::Reshape(R1D.ReadWrite(), test_element::ndof, num_elements);
    mfem::DeviceTensor<4, const double> J_d       = mfem::Reshape(J1D.Read(), q * q * q, dim, dim, num_elements);
    int                                 blocksize = 128;
    int                                 gridsize  = (num_elements * q * q * q + blocksize - 1) / blocksize;
    double                              runtime   = time([&]() {
      for (int i = 0; i < num_runs; i++) {
        serac::reference_cuda_kernel<Geometry::Hexahedron, test, trial, q>
            <<<gridsize, blocksize>>>(u_d, r_d, J_d, num_elements, qfunc);
        compiler::please_do_not_optimize_away(&R1D);
      }
      hipDeviceSynchronize();
    });
    std::cout << "average refernce kernel time: " << runtime / num_runs << std::endl;
  }
  auto answer_reference = R1D;

  {
    R1D = 0.0;

    auto          rule = serac::MakeGaussLegendreRule<Geometry::Hexahedron, q>();
    constexpr int epb  = serac::elements_per_block<Geometry::Hexahedron>(q);
    dim3          blocksize{q * q * q, epb, 1};
    int           gridsize = (num_elements + epb - 1) / epb;
    double        runtime  = time([&]() {
      for (int i = 0; i < num_runs; i++) {
        serac::batched_cuda_kernel<Geometry::Hexahedron, test, trial, q>
            <<<gridsize, blocksize>>>(U1D.Read(), R1D.ReadWrite(), J1D.Read(), rule, num_elements, qfunc);
        compiler::please_do_not_optimize_away(&R1D);
      }
      hipDeviceSynchronize();
    });
    std::cout << "average batched kernel time: " << runtime / num_runs << std::endl;
  }
  auto error = answer_reference;
  error -= R1D;
  double relative_error = error.Norml2() / answer_reference.Norml2();
  std::cout << "error: " << relative_error << std::endl;

  {
    R1D            = 0.0;
    bool symmetric = false;

    // I think the "o" and "c" are supposed to be short for
    // "open" and "closed", referring to placing interpolation
    // nodes at the gauss-legendre, and gauss-lobatto points (respectively)
    mfem::Array<double> bo_((n - 1) * q);
    mfem::Array<double> bc_(n * q);
    mfem::Array<double> bot_((n - 1) * q);
    mfem::Array<double> bct_(n * q);
    mfem::Array<double> gc_(n * q);
    mfem::Array<double> gct_(n * q);

    auto Bo  = mfem::Reshape(bo_.HostReadWrite(), q, n - 1);
    auto Bc  = mfem::Reshape(bc_.HostReadWrite(), q, n);
    auto Bot = mfem::Reshape(bot_.HostReadWrite(), n - 1, q);
    auto Bct = mfem::Reshape(bct_.HostReadWrite(), n, q);
    auto Gc  = mfem::Reshape(gc_.HostReadWrite(), q, n);
    auto Gct = mfem::Reshape(gct_.HostReadWrite(), n, q);

    for (int i = 0; i < q; i++) {
      auto lobatto_value      = serac::GaussLobattoInterpolation<n>(rule.points_1D[i]);
      auto lobatto_derivative = serac::GaussLobattoInterpolationDerivative<n>(rule.points_1D[i]);

      for (int j = 0; j < n; j++) {
        Bct(j, i) = Bc(i, j) = lobatto_value[j];
        Gct(j, i) = Gc(i, j) = lobatto_derivative[j];
      }

      auto legendre_value = serac::GaussLegendreInterpolation<n - 1>(rule.points_1D[i]);
      for (int j = 0; j < n - 1; j++) {
        Bot(j, i) = Bo(i, j) = legendre_value[j];
      }
    }

    double mass_runtime = time([&]() {
      for (int i = 0; i < num_runs; i++) {
        mfem::SmemPAHcurlMassApply3D<n, q>(n, q, num_elements, symmetric = false, bo_, bc_, bot_, bct_,
                                           rho_invJ_invJT_dv_1D, U1D, R1D);
        compiler::please_do_not_optimize_away(&R1D);
      }
    });
    std::cout << "average mfem mass kernel time: " << mass_runtime / num_runs << std::endl;

    double curlcurl_runtime = time([&]() {
      for (int i = 0; i < num_runs; i++) {
        mfem::SmemPACurlCurlApply3D<n, q>(n, q, symmetric = false, num_elements, bo_, bc_, bot_, bct_, gc_, gct_,
                                          k_JTJ_dv_over_detJsq_1D, U1D, R1D);
        compiler::please_do_not_optimize_away(&R1D);
      }
    });
    std::cout << "average mfem curlcurl kernel time: " << curlcurl_runtime / num_runs << std::endl;

    std::cout << "average mfem combined kernel time: " << (mass_runtime + curlcurl_runtime) / num_runs << std::endl;
  }
  auto answer_mfem = R1D;
  error            = answer_reference;
  error -= answer_mfem;
  relative_error = error.Norml2() / answer_reference.Norml2();
  std::cout << "error: " << relative_error << std::endl;
}

int main()
{
  mfem::Device device("cuda");

  int num_runs     = 10;
  int num_elements = 20000;
  h1_h1_test_2D<1 /* polynomial order */, 2 /* quadrature points / dim */>(num_elements, num_runs);
  h1_h1_test_2D<2 /* polynomial order */, 2 /* quadrature points / dim */>(num_elements, num_runs);
  h1_h1_test_2D<3 /* polynomial order */, 2 /* quadrature points / dim */>(num_elements, num_runs);
  h1_h1_test_3D<1 /* polynomial order */, 2 /* quadrature points / dim */>(num_elements, num_runs);
  h1_h1_test_3D<2 /* polynomial order */, 3 /* quadrature points / dim */>(num_elements, num_runs);
  h1_h1_test_3D<3 /* polynomial order */, 4 /* quadrature points / dim */>(num_elements, num_runs);
  hcurl_hcurl_test_2D<1 /* polynomial order */, 2 /* quadrature points / dim */>(num_elements, num_runs);
  hcurl_hcurl_test_2D<2 /* polynomial order */, 3 /* quadrature points / dim */>(num_elements, num_runs);
  hcurl_hcurl_test_2D<3 /* polynomial order */, 4 /* quadrature points / dim */>(num_elements, num_runs);
  hcurl_hcurl_test_3D<1 /* polynomial order */, 2 /* quadrature points / dim */>(num_elements, num_runs);
  hcurl_hcurl_test_3D<2 /* polynomial order */, 3 /* quadrature points / dim */>(num_elements, num_runs);
  hcurl_hcurl_test_3D<3 /* polynomial order */, 4 /* quadrature points / dim */>(num_elements, num_runs);
}
